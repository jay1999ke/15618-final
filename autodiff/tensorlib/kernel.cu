#include "hip/hip_runtime.h"
#include "tensor.h"

__global__ void _add(float *a, float *b, float *res, int dim0, int dim1) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim0 * dim1) {
        res[idx] = a[idx] + b[idx];
    }
}

__global__ void _mul(float *a, float *b, float *res, int dim0, int dim1) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < dim0 * dim1) {
        res[idx] = a[idx] * b[idx];
    }
}

__global__ void _sum(float *a, float *res, int dim0, int dim1, int axis) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int res_dim0, res_dim1;
    if (axis == 0) {
        res_dim0 = 1;
        res_dim1 = dim1;
    } else {
        res_dim0 = dim0;
        res_dim1 = 1;
    }

    if (idx < res_dim0 * res_dim1) {
        float temp_result = 0;
        if (axis == 0) {
            for (int i = 0; i < dim0; i++) {
                temp_result += a[i * dim1 + idx];
            }
        } else {
            for (int j = 0; j < dim1; j++) {
                temp_result += a[idx * dim1 + j];
            }
        }
        res[idx] = temp_result;
    }
}

__global__ void _bct(float *a, float *res, int res_dim0, int res_dim1,
                     int axis) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < res_dim0 * res_dim1) {
        if (axis == 0)
            res[idx] = a[idx % res_dim1];
        else {
            res[idx] = a[idx / res_dim1];
        }
    }
}

__global__ void _exp(float *a, float *res, int dim0, int dim1) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dim0 * dim1) {
        res[idx] = exp(a[idx]);
    }
}

__global__ void _tsp(float *a, float *res, int dim0, int dim1) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int i = idx % dim1;
    unsigned int j = idx / dim1;

    if (idx < dim0 * dim1) {
        res[j + i * dim0] = a[idx];
    }
}

__global__ void _pow(float *a, float *res, int dim0, int dim1, float val) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dim0 * dim1) {
        res[idx] = pow(a[idx], val);
    }
}

__global__ void _matmul(float *a, float *b, float *res, int dim0_a, int dim1_a,
                        int dim1_b) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < dim0_a * dim1_b) {
        int row = idx / dim1_b;
        int col = idx % dim1_b;
        float sum = 0.0;

        for (int i = 0; i < dim1_a; i++) {
            sum += a[row * dim1_a + i] * b[i * dim1_b + col];
        }

        res[idx] = sum;
    }
}