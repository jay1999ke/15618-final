#include "tensor.h"

Tensor *createGPUTensor(size_t rows, size_t cols) {
    Tensor *tensor = new Tensor(rows, cols);
    tensor->_gpu();
    return tensor;
}

void Tensor::setOnGpu(bool val) { on_gpu = val; }

void Tensor::_gpu() {
    if (gpu_data == nullptr)
        hipMalloc(&gpu_data, size());
    setOnGpu(true);
}

void Tensor::gpu() {
    _gpu();
    hipMemcpy(dataGpu(), data(), size(), hipMemcpyHostToDevice);
}

void Tensor::cpu() {
    if (dataGpu() != nullptr)
        hipMemcpy(data(), dataGpu(), size(), hipMemcpyDeviceToHost);
}

void Tensor::gpuFree() {
    if (dataGpu() != nullptr)
        hipFree(dataGpu());
    gpu_data = nullptr;
    setOnGpu(false);
}

void gpu_set_zero(Tensor *a) {
    hipMemset(a->dataGpu(), 0, a->size());
}

Tensor *gpu_add(Tensor *a, Tensor *b) {
    a->onGpuAssert();
    b->onGpuAssert();
    a->sameShapeAssert(b);

    py::buffer_info a_info = a->request();
    py::buffer_info b_info = b->request();

    int dim0 = a_info.shape[0];
    int dim1 = a_info.shape[1];

    Tensor *result = createGPUTensor(dim0, dim1);

    const int threadsPerBlock = 512;
    int blocks = (result->size() + threadsPerBlock - 1) / threadsPerBlock;

    _add<<<blocks, threadsPerBlock>>>(a->dataGpu(), b->dataGpu(),
                                      result->dataGpu(), dim0, dim1);

    return result;
}

Tensor *gpu_mul(Tensor *a, Tensor *b) {
    a->onGpuAssert();
    b->onGpuAssert();
    a->sameShapeAssert(b);
    
    py::buffer_info a_info = a->request();
    py::buffer_info b_info = b->request();

    int dim0 = a_info.shape[0];
    int dim1 = a_info.shape[1];

    Tensor *result = createGPUTensor(dim0, dim1);

    const int threadsPerBlock = 512;
    int blocks = (result->size() + threadsPerBlock - 1) / threadsPerBlock;

    _mul<<<blocks, threadsPerBlock>>>(a->dataGpu(), b->dataGpu(),
                                      result->dataGpu(), dim0, dim1);

    return result;
}

Tensor *gpu_sum(Tensor *a, int axis) {
    a->onGpuAssert();

    py::buffer_info a_info = a->request();

    int dim0 = a_info.shape[0];
    int dim1 = a_info.shape[1];

    int res_dim0, res_dim1;

    if (axis == 0) {
        res_dim0 = 1;
        res_dim1 = dim1;
    } else if (axis == 1) {
        res_dim0 = dim0;
        res_dim1 = 1;
    } else {
        throw std::runtime_error("Invalid sum axis");
    }

    Tensor *result = createGPUTensor(res_dim0, res_dim1);

    const int threadsPerBlock = 512;
    int blocks = (result->size() + threadsPerBlock - 1) / threadsPerBlock;

    _sum<<<blocks, threadsPerBlock>>>(a->dataGpu(), result->dataGpu(), dim0,
                                      dim1, axis);

    return result;
}

Tensor *gpu_bct(Tensor *a, int axis, int dim) {
    a->onGpuAssert();

    py::buffer_info a_info = a->request();

    int dim0 = a_info.shape[0];
    int dim1 = a_info.shape[1];

    int res_dim0, res_dim1;

    if (axis == 0) {
        res_dim0 = dim;
        res_dim1 = dim1;
    } else if (axis == 1) {
        res_dim0 = dim0;
        res_dim1 = dim;
    } else {
        throw std::runtime_error("Invalid sum axis");
    }

    Tensor *result = createGPUTensor(res_dim0, res_dim1);

    const int threadsPerBlock = 512;
    int blocks = (result->size() + threadsPerBlock - 1) / threadsPerBlock;

    _bct<<<blocks, threadsPerBlock>>>(a->dataGpu(), result->dataGpu(), res_dim0,
                                      res_dim1, axis);

    return result;
}